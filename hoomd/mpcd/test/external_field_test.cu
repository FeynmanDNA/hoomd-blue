#include "hip/hip_runtime.h"
#include "external_field_test.cuh"
#include "hoomd/GPUPolymorph.cuh"
#include <stdio.h>

namespace kernel
{
__global__ void test_field(Scalar3* out, const mpcd::ExternalField* field, const Scalar3* pos, const unsigned int N)
    {
    const int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= N) return;
    out[idx] = field->evaluate(pos[idx]);
    }
}

hipError_t test_field(Scalar3* out, const mpcd::ExternalField* field, const Scalar3* pos, const unsigned int N)
    {
    const unsigned int block_size = 32;
    const unsigned int num_blocks = (N + block_size - 1)/block_size;
    kernel::test_field<<<num_blocks,block_size>>>(out, field, pos, N);
    return hipSuccess;
    }

// if the instantiation is also done here, then everything works fine?
//template mpcd::ConstantForce* hoomd::gpu::device_new(Scalar3);
